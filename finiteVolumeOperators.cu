#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
using namespace std;
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Divergence of a Vector with variable coefficient- term in momentum eqn   !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
__global__ void DivGPU(double* Dn, double* Phi, double* U, double* V, int row, int col,double delX,double delY){
  // Get global thread ID
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int c = k%col;
  int r = k/col;
  if(c>0 && c<(col-1) && r>0 && r<(row-1)){


   double PhiP = Phi[k];
   double PhiE = Phi[k+1];
   double PhiW = Phi[k-1];
   double PhiN = Phi[k-col];
   double PhiS = Phi[k+col];

   double UP = U[k];
   double UE = U[k+1];
   double UW = U[k-1];
   //double UN = U[k-col];
   //double US = U[k+col];

   double VP = V[k];
   //double VE = V[k+1];
   //double VW = V[k-1];
   double VN = V[k-col];
   double VS = V[k+col];

   double Ee  = 0.5*(UE*PhiE+UP*PhiP);
   double Ew  = 0.5*(UW*PhiW+UP*PhiP);
   double Fn  = 0.5*(VN*PhiN+VP*PhiP);
   double Fs  = 0.5*(VS*PhiS+VP*PhiP);
   Dn[k]      = delX*(Fn-Fs)+delY*(Ee-Ew);
//{printf("Hi:DivGPU:");}
}
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Divergence of a Vector with No-coefficient- in continuity & source term  !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!

__global__ void DivergenceGPU(double* Dn, double* U, double* V,int row, int col, double delX, double delY){

  // Get global thread ID
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int c = k%col;
  int r = k/col;
  if(c>0 && c<(col-1) && r>0 && r<(row-1)){

   double UP = U[k];
   double UE = U[k+1];
   double UW = U[k-1];
   //double UN = U[k-col];
   //double US = U[k+col];

   double VP = V[k];
   //double VE = V[k+1];
   //double VW = V[k-1];
   double VN = V[k-col];
   double VS = V[k+col];

   double Ue = 0.5*(UE+UP);
   double Uw = 0.5*(UW+UP);
   //double Un = 0.5*(UN+UP);
   //double Us = 0.5*(US+UP);

   //double Ve = 0.5*(VE+VP);
   //double Vw = 0.5*(VW+VP);
   double Vn = 0.5*(VN+VP);
   double Vs = 0.5*(VS+VP);

  Dn[k] = (Ue-Uw)*delY+(Vn-Vs)*delX;
  
 }
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Laplacian of a Scalar                                                    !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
__global__ void LaplacianGPU(double* Ln, double *Phi, int row, int col, double delX, double delY){
  // Get global thread ID
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int c = k%col;
  int r = k/col;

  // Do for only inner points
  if(c>0 && c<(col-1) && r>0 && r<(row-1)){
   double PhiP = Phi[k];
   double PhiE = Phi[k+1];
   double PhiW = Phi[k-1];
   double PhiN = Phi[k-col];
   double PhiS = Phi[k+col];

   double Ee  = (PhiE-PhiP)/delX;
   double Ew  = (PhiP-PhiW)/delX;
   double Fn  = (PhiN-PhiP)/delY;
   double Fs  = (PhiP-PhiS)/delY;
   Ln[k]      = delX*(Fn-Fs)+delY*(Ee-Ew);


//{printf("Hi:Laplacian %6.3f\n",Ee);}

}
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Gradient                                                                 !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
__global__ void gradientGPU(double* gradxPhi1,double* gradyPhi1,double* Phi,
                        int row, int col, double delX, double delY){
  // Get global thread ID
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int c = k%col;
  int r = k/col;
  
  // Do for only inner points
  if(c>0 && c<(col-1) && r>0 && r<(row-1)){
   double PhiE = Phi[k+1];
   double PhiW = Phi[k-1];
   double PhiN = Phi[k-col];
   double PhiS = Phi[k+col];
   double PhiP = Phi[k];

   double Phie = 0.5*(PhiE + PhiP);
   double Phiw = 0.5*(PhiW + PhiP);
   double Phin = 0.5*(PhiN + PhiP);
   double Phis = 0.5*(PhiS + PhiP);

   gradxPhi1[k] = (Phie-Phiw)/delX;
   gradyPhi1[k] = (Phin-Phis)/delY;
 
//{printf("Hi:gradient:%6.4f:",gradxPhi1[k]);}

 }
}

//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Divergence of a Vector with variable coefficient- term in momentum eqn   !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
void Div(double* Dn, double* Phi, double* U, double* V, int row, int col,double delX,double delY){

for(int i = 1; i<(row-1); ++i){
 for(int j =1; j<(col-1); ++j){
      int k    = i*col+j;
   double PhiP = Phi[k];
   double PhiE = Phi[k+1];
   double PhiW = Phi[k-1];
   double PhiN = Phi[k-col];
   double PhiS = Phi[k+col];

   double UP = U[k];
   double UE = U[k+1];
   double UW = U[k-1];
   //double UN = U[k-col];
  // double US = U[k+col];

   double VP = V[k];
  // double VE = V[k+1];
  // double VW = V[k-1];
   double VN = V[k-col];
   double VS = V[k+col];

   double Ee  = 0.5*(UE*PhiE+UP*PhiP);
   double Ew  = 0.5*(UW*PhiW+UP*PhiP);
   double Fn  = 0.5*(VN*PhiN+VP*PhiP);
   double Fs  = 0.5*(VS*PhiS+VP*PhiP);
   Dn[k]      = delX*(Fn-Fs)+delY*(Ee-Ew);
      }
   }
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Divergence of a Vector with No-coefficient- in continuity & source term  !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!

void Divergence(double* Dn, double* U, double* V,int row, int col, double delX, double delY){

for(int i = 1; i<(row-1); ++i){
 for(int j =1; j<(col-1); ++j){
       int k = i*col+j;
   double UP = U[k];
   double UE = U[k+1];
   double UW = U[k-1];
   //double UN = U[k-col];
   //double US = U[k+col];

   double VP = V[k];
   //double VE = V[k+1];
   //double VW = V[k-1];
   double VN = V[k-col];
   double VS = V[k+col];

   double Ue = 0.5*(UE+UP);
   double Uw = 0.5*(UW+UP);
   //double Un = 0.5*(UN+UP);
   //double Us = 0.5*(US+UP);

   //double Ve = 0.5*(VE+VP);
   //double Vw = 0.5*(VW+VP);
   double Vn = 0.5*(VN+VP);
   double Vs = 0.5*(VS+VP);

  Dn[k] = (Ue-Uw)*delY+(Vn-Vs)*delX;
   }
 }
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Laplacian of a Scalar                                                    !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
void Laplacian(double* Ln, double *Phi, int row, int col, double delX, double delY){
for(int i = 1; i<(row-1); i++){
 for(int j =1; j<(col-1); j++){
   int k = i*col+j;
   double PhiP = Phi[k];
   double PhiE = Phi[k+1];
   double PhiW = Phi[k-1];
   double PhiN = Phi[k-col];
   double PhiS = Phi[k+col];

   double Ee  = (PhiE-PhiP)/delX;
   double Ew  = (PhiP-PhiW)/delX;
   double Fn  = (PhiN-PhiP)/delY;
   double Fs  = (PhiP-PhiS)/delY;
   Ln[k]      = delX*(Fn-Fs)+delY*(Ee-Ew);
     }
  }
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
// Gradient                                                                 !
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++!
void gradient(double* gradxPhi,double* gradyPhi,double* Phi,
                        int row, int col, double delX, double delY){
for(int i = 1; i<(row-1); ++i){
 for(int j =1; j<(col-1); ++j){

   int       k = i*col+j;
   double PhiE = Phi[k+1];
   double PhiW = Phi[k-1];
   double PhiN = Phi[k-col];
   double PhiS = Phi[k+col];
   double PhiP = Phi[k];

   double Phie = 0.5*(PhiE + PhiP);
   double Phiw = 0.5*(PhiW + PhiP);
   double Phin = 0.5*(PhiN + PhiP);
   double Phis = 0.5*(PhiS + PhiP);

   gradxPhi[k] = (Phie-Phiw)/delX;
   gradyPhi[k] = (Phin-Phis)/delY;
    }
  }
}

